#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>

void calculateCpuMax(float* x, int N, float* counter) {
	float max = x[0];
	for (int i = 0; i < N; i++) {
		if (x[i] > max && i % 2 == 0) {
			max = x[i];
		}
	}
	*counter = max;
}

__global__ void calculateGpuMax(int n, float* x, float* counter)
{
	int threadCount = 1000;
	int index = threadIdx.x;
	int startIndex = index * threadCount;
	int endIndex = (index + 1) * threadCount;

	float max = x[startIndex];
	for (int i = startIndex + 1; i < endIndex && i < n; i++) {
		if (x[i] > max && i % 2 == 0) {
			max = x[i];
		}
	}
	counter[index] = max;
}

int main(void)
{
	int N = 100000000;

	float *array_h;
	float* cpuMax = (float*)malloc(sizeof(float));

	clock_t start, end;
	float timeElapsed;
	float timeElapsedGpu;

	printf("Number of elements = %i\n", N);

	srand(time(NULL));

	array_h = (float*)malloc(N * sizeof(float));

	for (int i = 0; i < N; i++) {
		array_h[i] = rand() % 256;
	}

	start = clock();

	*cpuMax = 0;

	calculateCpuMax(array_h, N, cpuMax);

	end = clock();

	timeElapsed = (double)(end - start) / CLOCKS_PER_SEC * 1000;

	printf("CPU max even = %.0f\n", *cpuMax);
	printf("CPU time = %.3fms\n", timeElapsed);

	float *counter_h;
	float *array_d;
	float *counter_d;

	int threadCount = 1000;

	counter_h = (float*)malloc(threadCount * sizeof(float));

	hipMalloc(&array_d, N * sizeof(float));
	hipMalloc(&counter_d, threadCount * sizeof(float));

	*counter_h = 0;

	hipMemcpy(array_d, array_h, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(counter_d, counter_h, threadCount * sizeof(float), hipMemcpyHostToDevice);

	start = clock();

	calculateGpuMax << <1, threadCount >> >(N, array_d, counter_d);

	hipDeviceSynchronize();

	hipMemcpy(counter_h, counter_d, threadCount * sizeof(float), hipMemcpyDeviceToHost);

	float gpuMax = counter_h[0];

	for (int i = 1; i < threadCount; i++) {
		if (counter_h[i] > gpuMax) {
			gpuMax = counter_h[i];
		}
	}

	free(counter_h);

	hipFree(array_d);
	hipFree(counter_d);

	end = clock();

	timeElapsedGpu = (double)(end - start) / CLOCKS_PER_SEC * 1000;

	printf("GPU max even = %0.f\n", gpuMax);
	printf("GPU time = %.3fms\n", timeElapsedGpu);

	printf("Speed Coefficient = %.3f\n", timeElapsed / timeElapsedGpu);

	free(array_h);
	free(cpuMax);

	system("pause");

	return 0;
}

