#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <random>
#include <Windows.h>
#include <stdlib.h>
using namespace std;

__global__ void Addition(double* answer, double* mass1, double* mass2, int* n)
{
	int row = blockIdx.x * 10000 + threadIdx.x;
	int column = blockIdx.y * 10000 + threadIdx.y;
	int border = n[0] * n[0];
	answer[row * n[0] + column] = 0;
	for (int p = 0; p < n[0]; p++)
	{
		answer[row * n[0] + column] = (mass1[row * n[0] + p] + mass2[p * n[0] + column]);
	}
}

double* Addition_CPU(double* mass1, double* mass2, int* n)
{
	double* answer = new double[n[0] * n[0]];
	for (int p = 0; p < n[0]; p++)
	{
		for (int q = 0; q < n[0]; q++)
		{
			answer[p * n[0] + q] = 0;
			for (int r = 0; r < n[0]; r++)
			{
				answer[p * n[0] + q] = (mass1[(p * n[0] + r)] + mass2[r * n[0] + q]);
			}
		}
	}
	return answer;
}

void main()
{
	int e = 1;
	int start;
	int* n = new int[1];
	n[0] = 1000;
	srand((unsigned)time(NULL));
	//printf("Generating first matr\n");
	start = GetTickCount();
	double* mass1 = new double[n[0] * n[0]];
	for (int p = 0; p < n[0] * n[0]; p++)
	{
		mass1[p] = (double)rand() / (double)rand();
	}
	//printf("Matr 1 ready\n");
	//printf("Generating time: %i\n", GetTickCount() - start);
	//printf("\nGenerating second massive\n");
	double* mass2 = new double[n[0] * n[0]];
	start = GetTickCount();
	for (int p = 0; p < n[0] * n[0]; p++)
	{
		mass2[p] = (double)rand() / (double)rand();
	}
	//printf("Matr 2 ready\n");
	//printf("Generating time: %i\n\n", GetTickCount() - start);

	printf("CPU working\n");
	start = GetTickCount();
	double* answer_CPU = Addition_CPU(mass1, mass2, n);
	int CPU_time = GetTickCount() - start;
	printf("CPU compute time: %i\n\n", CPU_time);


	printf("GPU working\n");
	start = GetTickCount();
	double* cuda_answer;
	hipMalloc(&cuda_answer, sizeof(double) * n[0] * n[0]);
	double* cuda_mass1;
	hipMalloc(&cuda_mass1, sizeof(double) * n[0] * n[0]);
	hipMemcpy(cuda_mass1, mass1, sizeof(double) * n[0] * n[0], hipMemcpyHostToDevice);
	double* cuda_mass2;
	hipMalloc(&cuda_mass2, sizeof(double) * n[0] * n[0]);
	hipMemcpy(cuda_mass2, mass2, sizeof(double) * n[0] * n[0], hipMemcpyHostToDevice);
	int* cuda_n;
	hipMalloc(&cuda_n, sizeof(int));
	hipMemcpy(cuda_n, n, sizeof(int), hipMemcpyHostToDevice);
	double* answer = new double[n[0] * n[0]];

	Addition << <1,1000 >> >(cuda_answer, cuda_mass1, cuda_mass2, cuda_n);
	hipDeviceSynchronize();
	hipMemcpy(answer, cuda_answer, sizeof(double) * n[0] * n[0], hipMemcpyDeviceToHost);
	int GPU_time = GetTickCount() - start;
	printf("GPU compute time: %i\n", GPU_time);


	bool correct = true;
	for (int p = 0; p < n[0] * n[0]; p++)
	{
		if (abs(answer[p] - answer_CPU[p] > e))
		{
			correct = false;
			break;
		}
	}
	if (correct)
	{
		printf("\nAnswers are equal\n");
	}
	else
	{
		printf("\nAnswers aren't equal\n");
	}
	printf("\nCoefficient: %f\n", ((double)CPU_time / (double)GPU_time));
	scanf("%d");
}
